
#include <hip/hip_runtime.h>
// Colour Sine wave Kernal
// Based on kernal_colour in kernelVBO.cpp by Rob Farber
__global__ void kernel(float4* dVertexArray, uchar4 *dColorArray,
           unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // Each thread is unique point (u,v) in interval [-1,1],[-1,1]
    const float u = 2.0* (x/(float)width)  - 1.0f;
    const float v = 2.0* (y/(float)height) - 1.0f;
    const float w = 0.5 * sinf(4.0*u + time) * cosf(4.0*v + time);

    // Update vertex array for point
    dVertexArray[y*width+x] = make_float4(u, w, v, 1.0f);

    // Update colour array for point
    dColorArray[y*width+x].w = 0;
    dColorArray[y*width+x].x = 255.f *0.5*(1.f+sinf(w+x));
    dColorArray[y*width+x].y = 255.f *0.5*(1.f+sinf(x)*cosf(y));
    dColorArray[y*width+x].z = 255.f *0.5*(1.f+sinf(w+time/10.f));
}

extern "C" void launch_kernel(float4* dVertexArray, uchar4* dColourArray,
            unsigned int width, unsigned int height, float time)
{
  dim3 block(8, 8, 1);
  dim3 grid(width / block.x, height / block.y, 1);
  kernel<<< grid, block>>>(dVertexArray, dColourArray, width, height, time);
}
